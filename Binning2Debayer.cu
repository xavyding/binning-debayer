#include "hip/hip_runtime.h"
#include "Binning2Debayer.h"

static constexpr int KERNEL_BLOCK_SIZE = 16;

__global__ void __gpuKernelv2_Binning2demosaicingC3x16(const ushort* inputData, ushort* outputData, const int outputRows, const int outputCols, \
    int blockSkipFactor, const size_t inputStep, const size_t outputStep, const int inputChannels, const int outputChannels = 3) {
    // 16 bits version
    /* Input mosaic color bloc representation:
    +-------+   x (inputOffsetX)
    | R | Gr| 
    |---+---+
    | Gb| B |
    +---+---+
    
    y (inputOffsetY)
    */
    if (blockIdx.x * blockSkipFactor >= outputCols || blockIdx.y * blockSkipFactor >= outputRows) return;

    int inputPosX = threadIdx.x / 2;
    int inputOffsetX = threadIdx.x % 2;
    int inputPosY = threadIdx.y / 2;
    int inputOffsetY = threadIdx.y % 2;

    if (inputOffsetX == 1 && inputOffsetY == 0) return;

    int outputIdx = (inputPosY + blockIdx.y * blockSkipFactor) * outputStep + (inputPosX + blockIdx.x * blockSkipFactor) * outputChannels;
    int inputIdx = (threadIdx.y + blockIdx.y * blockDim.y) * inputStep + (threadIdx.x + blockIdx.x * blockDim.x) * inputChannels;

    if (inputOffsetX == 1 && inputOffsetY == 1) {  // Blue
        outputData[outputIdx] = inputData[inputIdx];
    } else if (inputOffsetX == 0 && inputOffsetY == 1) {  // Green (Gb)
        outputData[outputIdx + 1] = inputData[inputIdx];
    } else if (inputOffsetX == 0 && inputOffsetY == 0) {  // Red
        outputData[outputIdx + 2] = inputData[inputIdx];
    } else {
        return;
    }
}


__host__ void Binning2Debayer::demosaicing(const cv::cuda::GpuMat & input, cv::cuda::GpuMat & output) {
    // check if already 3 channels, in which case no need to do demosaicing.
    if (input.channels() == 3) {
        output = input;
        return;
    }

    int blockSize = KERNEL_BLOCK_SIZE;

    dim3 blocksPerGrid = dim3(static_cast<int>(input.step/sizeof(ushort)/blockSize), static_cast<int>(input.rows/blockSize));
    dim3 threadsPerBlock = dim3(blockSize, blockSize);
    __gpuKernelv2_Binning2demosaicingC3x16<<<blocksPerGrid, threadsPerBlock>>> (input.ptr<ushort>(), output.ptr<ushort>(), \
        output.rows, output.cols, blockSize/_binningFactor, input.step/sizeof(ushort), output.step/sizeof(ushort),  1, 3);
}

